
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel declarations
extern __global__ void helloWorldKernel1();
extern __global__ void helloWorldKernel2();

int main() {
    // Launch kernel 1
    printf("Launching helloWorldKernel1...\n");
    helloWorldKernel1<<<1, 256>>>();
    hipDeviceSynchronize();  // Wait for kernel 1 to complete

    // Launch kernel 2
    printf("\nLaunching helloWorldKernel2...\n");
    helloWorldKernel2<<<1, 256>>>();
    hipDeviceSynchronize();  // Wait for kernel 2 to complete

    return 0;
}
